#include "hip/hip_runtime.h"
#include "../include/wmma_spmm.cuh"
#include "spmm_utils/dense_tile.h"
#include "spmm_utils/sparse_tile.h"
#include "spmm_utils/compute_utils.h"
#include "spmm_utils/output_tile.h"
#include <stdio.h>
#include <mma.h>

using namespace nvcuda;

namespace spmm{

//4-bit Tile_N = 128 with 2 warps
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_4b(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    // Each int32 has 8 4-bit integers with double buffers
    __shared__ int values_tile_array[Tile_K*VecLength/4];
    __shared__ int column_indices_tile_array[Tile_K*2];

    //padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has eight 4-bit integers
    wmmaSparseTile_4b<LoadType, VecType, Tile_K * VecLength / 8, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_4b<Tile_K * VecLength / 8> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_4b<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//8-bit Tile_N = 128 with 4 warps
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_8b(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength/2];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has four 8-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/4 + 8*7];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has four 8-bit integers
    wmmaSparseTile_8b<LoadType, VecType, Tile_K * VecLength / 4, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_8b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/4, dimN_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_8b<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_8b<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//16-bit 8-bit Tile_N = 128 with 4 warps
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b8b(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has four 8-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/4 + 8*7];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has two 16-bit integers
    wmmaSparseTile_16b8b<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_8b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/4, dimN_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_16b8b<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b8b<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}


//16-bit 8-bit Tile_N = 128 with 4 warps 8v
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b8b8v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has four 8-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/4 + 8*7];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has two 16-bit integers
    wmmaSparseTile_16b8b8v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_8b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/4, dimN_index/4, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_16b8b8v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b8b8v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix);
    output_tile_storer.Store();
}

//8-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_8b4b(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength/2];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_8b4b<LoadType, VecType, Tile_K * VecLength / 4, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_8b4b<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_8b4b<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//12-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_12b4b2v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices, double buffers
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_12b4b2v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_12b4b2v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_12b4b2v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//12-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_12b4b4v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices, double buffers
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_12b4b4v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_12b4b4v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_12b4b4v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix);
    output_tile_storer.Store();
}

//12-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_12b4b8v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    const int lane_size = blockDim.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices, double buffers
    __shared__ int values_tile_array[Tile_K*6]; //8v 12bit only requires Tile_k * 6
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_12b4b8v<LoadType, VecType, Tile_K * 3, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, lane_size, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_2[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_12b4b8v<Tile_K * 3> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, output_fragment_2, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_12b4b8v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_fragment_2, output_matrix);
    output_tile_storer.Store();
}

//16-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b4b2v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices, double buffers
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_12b4b2v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_12b4b2v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_12b4b2v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//16-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b4b4v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices, double buffers
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_16b4b4v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_16b4b4v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b4b4v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix);
    output_tile_storer.Store();
}

//16-bit A 4-bit B Tile_N = 128 warps = 2
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b4b8v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    const int lane_size = blockDim.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices, double buffers
    __shared__ int values_tile_array[Tile_K*VecLength]; //8v 12bit only requires Tile_k * 6
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_16b4b8v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, lane_size, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_2[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_3[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_16b4b8v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, output_fragment_2, output_fragment_3, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b4b8v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_fragment_2, output_fragment_3, output_matrix);
    output_tile_storer.Store();
}

//8-bit A 4-bit B Tile_N = 128 warps = 2, 8v
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_8b4b8v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength/2];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // each int value has four 4-bit values, padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/8 + 8*3];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    wmmaSparseTile_8b4b8v<LoadType, VecType, Tile_K * VecLength / 4, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[8] = {};
    // Initialize the pointers to the dense rhs matrix
    wmmaDenseTile_4b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/8, dimN_index/8, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    __align__(16) int output_fragment_0[Tile_N / Warps / 4] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 4] = {};
    wmmaComputeUtils_8b4b8v<Tile_K * VecLength / 4> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_8b4b8v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix);
    output_tile_storer.Store();
}


//16-bit 16-bit Tile_N = 64 with 4 warps
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has two 16-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/2 + 8*7];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has two 16-bit integers
    wmmaSparseTile_16b8b<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, threadIdx.x % 32, threadIdx.x / 32, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    // One int32 has two 16-bit integers
    wmmaDenseTile_16b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/2, dimN_index/2, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    // 16-bit decomposes into two 8-bits, x2
    __align__(16) int output_fragment[Tile_N / Warps / 2] = {};
    wmmaComputeUtils_16b<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment, output_matrix);
    output_tile_storer.Store();
}

//16-bit 16-bit Tile_N = 64 with 4 warps
template <typename LoadType, typename IndexType, typename VecType, 
          typename OutType, int Tile_K, 
          int Tile_N, int Warps, int VecLength>
__global__ void wmmaSpmm_kernel_16b8v(
    int m_vec, int dimN, int dimK, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    OutType* __restrict__ output_matrix)
{
    // For the wmma based implementation, we have Tile_M = 1
    int m_index_vec = blockIdx.x;
    int dimN_index = blockIdx.y * Tile_N;
    const int lane_id = threadIdx.x;
    // Threads that work on different m-dim indices are independent
    // If we're out of bounds in the m-dimension we can just return
    if (m_index_vec >= m_vec) return;
    m_index_vec = __ldg(row_indices + m_index_vec);

    // Load the row offset and calculate the number of nonzeros in the row
    int row_offset_vec = __ldg(row_offsets + m_index_vec*2);
    int nonzeros = __ldg(row_offsets + m_index_vec*2 + 1) - row_offset_vec;

    // Shared memory tiles for the lhs values and indices
    __shared__ int values_tile_array[Tile_K*VecLength];
    __shared__ int column_indices_tile_array[Tile_K*2];

    // One int32 has two 16-bit integers
    // Padding to avoid bank conflict 
    __shared__ int dense_tile_array[Tile_N*Tile_K/2 + 8*7];

    // Pointers to the shared memory tiles
    int* values_tile = values_tile_array;
    int* column_indices_tile = column_indices_tile_array;
    int* dense_tile = dense_tile_array;

    // Initialize the pointers to the sparse lhs matrix
    // One int32 has two 16-bit integers
    wmmaSparseTile_16b8b8v<LoadType, VecType, Tile_K * VecLength / 2, Tile_K> sparse_tile_loader(
        row_offset_vec, lane_id, values, column_indices,
        values_tile, column_indices_tile
    );

    __align__(16) int rhs_prefetch[4] = {};
    // Initialize the pointers to the dense rhs matrix
    // One int32 has two 16-bit integers
    wmmaDenseTile_16b<LoadType, Tile_K, Tile_N> dense_tile_loader(
        dimN/2, dimN_index/2, lane_id, rhs_matrix, column_indices_tile, dense_tile, rhs_prefetch 
    );

    // Accumulator registers for the output values.
    // Tile_N / warps / four threads in x-dim of output matrix
    // 16-bit decomposes into two 8-bits, x2
    __align__(16) int output_fragment_0[Tile_N / Warps / 2] = {};
    __align__(16) int output_fragment_1[Tile_N / Warps / 2] = {};
    wmmaComputeUtils_16b8v<Tile_K * VecLength / 2> computer(values_tile, dense_tile, output_fragment_0, output_fragment_1, lane_id);

    int steps = nonzeros / Tile_K;
    int residue = nonzeros % Tile_K;

    if(steps > 0){
        sparse_tile_loader.Load(0);
        __syncthreads();
        dense_tile_loader.Prefetch(0);

        int i = 1;
        #pragma unroll
        for(; i < steps; i++){
            dense_tile_loader.LoadRowfromRegister(i-1);
            sparse_tile_loader.Load(i);
            __syncthreads();
            dense_tile_loader.Prefetch(i);
            computer.TileMAC(i-1);
            __syncthreads();
        }

        dense_tile_loader.LoadRowfromRegister(i-1);
        __syncthreads();
        computer.TileMAC(i-1);
    }
   
    if(residue > 0){
        sparse_tile_loader.Residue();
        __syncthreads();
        dense_tile_loader.ResidueLoad(residue);
        __syncthreads();
        computer.TileMACResidue();
    } 

    wmmaOutputTile_16b8v<OutType> output_tile_storer(lane_id, VecLength, m_index_vec, dimN_index, dimN, output_fragment_0, output_fragment_1, output_matrix);
    output_tile_storer.Store();
}

template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_4b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    wmmaSpmm_kernel_4b<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
        m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);

    return hipGetLastError();
}

//4-bit Tile_N = 128 with 2 warps
hipError_t wmmaSpmm_4b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_4b_template<int, char, 1, 32, 128, 32, 2, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const char *>(values), rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_4b_template<int, short, 1, 32, 128, 32, 2, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const short *>(values), rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_4b_template<int, int, 1, 32, 128, 32, 2, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_8b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);

    wmmaSpmm_kernel_8b<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
        m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

//8-bit Tile_N = 128 with 4 warps
hipError_t wmmaSpmm_8b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_8b_template<int, short, 1, 16, 128, 32, 4, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const short *>(values), rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_8b_template<int, int, 1, 16, 128, 32, 4, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_8b_template<int, long long, 1, 16, 128, 32, 4, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

//12-bit 4-bit Tile_N = 128 with 2 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_12b4b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_12b4b8v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    if(vec_length == 4)
        wmmaSpmm_kernel_12b4b4v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    if(vec_length == 2)
        wmmaSpmm_kernel_12b4b2v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

hipError_t wmmaSpmm_12b4b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_12b4b_template<int, int, 1, 32, 128, 32, 2, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_12b4b_template<int, long long, 1, 32, 128, 32, 2, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_12b4b_template<int, long long, 1, 32, 128, 32, 2, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

//16-bit 4-bit Tile_N = 128 with 2 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_16b4b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_16b4b8v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    if(vec_length == 4)
        wmmaSpmm_kernel_16b4b4v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    if(vec_length == 2)
        wmmaSpmm_kernel_16b4b2v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

hipError_t wmmaSpmm_16b4b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_16b4b_template<int, int, 1, 32, 128, 32, 2, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_16b4b_template<int, long long, 1, 32, 128, 32, 2, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_16b4b_template<int, long long, 1, 32, 128, 32, 2, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

//8-bit 4-bit Tile_N = 128 with 2 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_8b4b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_8b4b8v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    else
        wmmaSpmm_kernel_8b4b<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

hipError_t wmmaSpmm_8b4b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_8b4b_template<int, short, 1, 32, 128, 32, 2, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const short *>(values), rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_8b4b_template<int, int, 1, 32, 128, 32, 2, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, values, rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_8b4b_template<int, long long, 1, 32, 128, 32, 2, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

//16-bit 8-bit Tile_N = 128 with 4 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_16b8b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_16b8b8v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    else
        wmmaSpmm_kernel_16b8b<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

hipError_t wmmaSpmm_16b8b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_16b8b_template<int, int, 1, 16, 128, 32, 4, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const int *>(values), rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_16b8b_template<int, long long, 1, 16, 128, 32, 4, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_16b8b_template<int, long long, 1, 16, 128, 32, 4, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

hipError_t wmmaSpmm_12b8b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_16b8b_template<int, int, 1, 16, 128, 32, 4, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const int *>(values), rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_16b8b_template<int, long long, 1, 16, 128, 32, 4, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_16b8b_template<int, long long, 1, 16, 128, 32, 4, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

//16-bit 16-bit Tile_N = 64 with 4 warps
template <typename IndexType, typename VecType, int Tile_M, int Tile_K, int Tile_N, int WarpWidth, int Warps, int VecLength>
hipError_t wmmaSpmm_16b_template(
    int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const VecType* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    dim3 grid_dim(ceil(static_cast<float>(m_vec) / Tile_M), ceil(static_cast<float>(n) / Tile_N), 1);
    dim3 block_dim(WarpWidth * Warps, Tile_M, 1);
    if(vec_length == 8)
        wmmaSpmm_kernel_16b8v<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    else
        wmmaSpmm_kernel_16b<int, int, VecType, int, Tile_K, Tile_N, Warps, VecLength><<<grid_dim, block_dim>>>(
            m_vec, n, k, row_indices, row_offsets, column_indices, values, rhs_matrix, output_matrix);
    return hipGetLastError();
}

hipError_t wmmaSpmm_16b(int m_vec, int vec_length, int n, int k, 
    const int* __restrict__ row_indices, 
    const int* __restrict__ row_offsets,
    const int* __restrict__ column_indices,
    const int* __restrict__ values,
    const int* __restrict__ rhs_matrix,
    int* __restrict__ output_matrix)
{
    switch(vec_length){
        case 2:
            return wmmaSpmm_16b_template<int, int, 1, 16, 64, 32, 4, 2>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const int *>(values), rhs_matrix, output_matrix);
            break;
        case 4:
            return wmmaSpmm_16b_template<int, long long, 1, 16, 64, 32, 4, 4>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        case 8:
            return wmmaSpmm_16b_template<int, long long, 1, 16, 64, 32, 4, 8>(m_vec, vec_length, n, k, row_indices, 
        		    row_offsets, column_indices, reinterpret_cast<const long long *>(values), rhs_matrix, output_matrix);
            break;
        default:
            printf("Unsupported Vector Length!\n");
            return hipGetLastError();
    }
}

}
